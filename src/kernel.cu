
#include <hip/hip_runtime.h>
extern "C" __global__ void addKernel(const float *a, const float *b, float *c,
                                     int N)
{
    int i = threadIdx.x;
    if (i >= N) return;

    c[i] = a[i] + b[i];
}